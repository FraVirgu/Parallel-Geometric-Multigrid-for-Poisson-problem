#include "Parallel_Method.cu"
#include "../Smoother.hpp"
class ParallelMultiGridSolver
{
private:
    int alpha;
    int N_final;
    int v1 = 1;
    int v2 = 1;

public:
    int N_cpu = 17;
    double *final_solution;
    double epsilon = 1e-7;

    explicit ParallelMultiGridSolver(int alpha_)
        : alpha(alpha_)
    {
    }

    void v_cycle(double *phi, double *f, int N, double h)
    {
        if (N <= N_cpu)
        {
            JacobiSmoother smoother(epsilon);
            MultigridSolver mg(&smoother, alpha, N);
            mg.v_cycle(phi, f, N, h);
            return;
        }
        else
        {
            Parallel device;
            device.ComputeJacobi(phi, f, N, N, h, v1);
            int L = N * N;
            double *res_dev;

            size_t bytes = L * sizeof(double);
            hipMallocManaged(&res_dev, bytes);
            hipDeviceSynchronize(); // Ensure memory is ready
            device.ComputeResidual(res_dev, phi, f, N, N, h);

            // Restrict residual to coarser grid
            int Nc = (N - 1) / 2 + 1;
            int Lc = Nc * Nc;

            double *res_coarse;
            size_t bytes_coarse = Lc * sizeof(double);
            hipMallocManaged(&res_coarse, bytes_coarse);
            hipDeviceSynchronize(); // Ensure memory is ready
            device.ComputeRestriction(res_dev, res_coarse, N, Nc);

            double *e_coarse;
            hipMallocManaged(&e_coarse, bytes_coarse);
            hipDeviceSynchronize(); // Ensure memory is ready
            v_cycle(e_coarse, res_coarse, Nc, 2 * h);

            device.ComputeProlungator(e_coarse, phi, Nc, N);
            device.ComputeJacobi(phi, f, N, N, h, v2);
        }
    }

    void w_cycle(double *phi, double *f, int N, double h)
    {
        if (N <= N_cpu)
        {
            JacobiSmoother smoother(epsilon);
            MultigridSolver mg(&smoother, alpha, N);
            mg.w_cycle(phi, f, N, h);
            return;
        }
        else
        {
            Parallel device;
            device.ComputeJacobi(phi, f, N, N, h, v1);
            int L = N * N;
            double *res_dev;

            size_t bytes = L * sizeof(double);
            hipMallocManaged(&res_dev, bytes);
            hipDeviceSynchronize(); // Ensure memory is ready
            device.ComputeResidual(res_dev, phi, f, N, N, h);

            // Restrict residual to coarser grid
            int Nc = (N - 1) / 2 + 1;
            int Lc = Nc * Nc;

            double *res_coarse;
            size_t bytes_coarse = Lc * sizeof(double);
            hipMallocManaged(&res_coarse, bytes_coarse);
            hipDeviceSynchronize(); // Ensure memory is ready
            device.ComputeRestriction(res_dev, res_coarse, N, Nc);

            double *e_coarse;
            hipMallocManaged(&e_coarse, bytes_coarse);
            hipDeviceSynchronize(); // Ensure memory is ready
            for (int i = 0; i < alpha; i++)
                w_cycle(e_coarse, res_coarse, Nc, 2 * h);

            device.ComputeProlungator(e_coarse, phi, Nc, N);
            device.ComputeJacobi(phi, f, N, N, h, v2);
        }
    }
};