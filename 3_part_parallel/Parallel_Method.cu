#include "hip/hip_runtime.h"
#include "iostream"
#include <hip/hip_runtime.h>
#include "../globals.hpp"
#include "../2_part_MG/MultiGrid.hpp"

__global__ void jacobi_kernel(double *x, double *f, int height, int width, double h_act)
{

    int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y_pos * width + x_pos;
    double x_out = 0.0;

    if (x_pos != 0 && y_pos != 0 && x_pos < width - 1 && y_pos < height - 1)
    {
        int index = y_pos * width + x_pos;
        x_out = 0.25 * ((h_act * h_act * f[index]) + x[index - 1] + x[index + 1] + x[index - width] + x[index + width]);
    }

    if (x_pos != 0 && y_pos != 0 && x_pos < width - 1 && y_pos < height - 1)
    {
        x[index] = x_out;
    }
}

__global__ void device_compute_residual(double *r, double *x, double *f, int height, int width, double h_act)
{
    int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y_pos * width + x_pos;
    double r_out = 0.0;

    if (x_pos != 0 && y_pos != 0 && x_pos < width - 1 && y_pos < height - 1)
    {
        r_out = f[index] - (1.0 / (h_act * h_act)) * (4 * x[index] - x[index - 1] - x[index + 1] - x[index - width] - x[index + width]);
    }

    __syncthreads();

    if (x_pos != 0 && y_pos != 0 && x_pos < width - 1 && y_pos < height - 1)
    {
        r[index] = r_out;
    }

    __syncthreads();
}

__global__ void restriction_kernel_full_weighting(const double *__restrict__ input, double *__restrict__ output,
                                                  int input_H, int input_W, int output_H, int output_W)
{
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_x <= 0 || out_x >= output_W - 1 || out_y <= 0 || out_y >= output_H - 1)
        return;

    int in_x = 2 * out_x;
    int in_y = 2 * out_y;

    int idx_out = out_y * output_W + out_x;

    // Index helpers
    int idx_center = in_y * input_W + in_x;
    int idx_left = idx_center - 1;
    int idx_right = idx_center + 1;
    int idx_top = idx_center - input_W;
    int idx_bottom = idx_center + input_W;
    int idx_tl = idx_top - 1;
    int idx_tr = idx_top + 1;
    int idx_bl = idx_bottom - 1;
    int idx_br = idx_bottom + 1;

    // Apply full-weighting stencil
    output[idx_out] =
        0.25 * input[idx_center] +
        0.125 * (input[idx_left] + input[idx_right] + input[idx_top] + input[idx_bottom]) +
        0.0625 * (input[idx_tl] + input[idx_tr] + input[idx_bl] + input[idx_br]);
}
__global__ void prolungator_kernel(const double *__restrict__ input, double *__restrict__ output,
                                   int input_H, int input_W, int output_H, int output_W)
{
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    if (y >= output_H || x >= output_W)
        return;

    // Don't touch boundary — Dirichlet condition
    if (y == 0 || y == output_H - 1 || x == 0 || x == output_W - 1)
    {
        output[y * output_W + x] = 0.0;
        return;
    }

    double value = 0.0;
    int cx = x / 2;
    int cy = y / 2;

    // Case 1: even-even (direct injection)
    if (x % 2 == 0 && y % 2 == 0)
    {
        value = input[cy * input_W + cx];
    }

    // Case 2: odd-odd (bilinear from 4 corners)
    else if (x % 2 == 1 && y % 2 == 1)
    {
        if (cx + 1 < input_W && cy + 1 < input_H)
        {
            value = 0.25 * (input[cy * input_W + cx] +
                            input[cy * input_W + (cx + 1)] +
                            input[(cy + 1) * input_W + cx] +
                            input[(cy + 1) * input_W + (cx + 1)]);
        }
    }

    // Case 3: even row, odd column (horizontal interpolation)
    else if (x % 2 == 1 && y % 2 == 0)
    {
        if (cx + 1 < input_W)
        {
            value = 0.5 * (input[cy * input_W + cx] +
                           input[cy * input_W + (cx + 1)]);
        }
    }

    // Case 4: odd row, even column (vertical interpolation)
    else if (x % 2 == 0 && y % 2 == 1)
    {
        if (cy + 1 < input_H)
        {
            value = 0.5 * (input[cy * input_W + cx] +
                           input[(cy + 1) * input_W + cx]);
        }
    }

    output[y * output_W + x] += value;
}

class Parallel
{

public:
    static void
    ComputeJacobi(double *d_x, double *d_f, int height, int weight, double h_act, int v)
    {
        int block_size = num_thread;
        int num_blocks = height / block_size;
        if (num_blocks == 0)
            num_blocks++;
        dim3 threadsPerBlock(block_size, block_size);
        dim3 numBlocks(num_blocks, num_blocks);
        for (int i = 0; i <= v; i++)
        {
            jacobi_kernel<<<numBlocks, threadsPerBlock>>>(d_x, d_f, height, weight, h_act);
            hipDeviceSynchronize();
        }

        return;
    }

    static void
    ComputeResidual(double *d_r, double *d_x, double *d_f, int height, int width, double h_act)
    {
        int block_size = num_thread;
        int num_blocks = height / block_size;
        if (num_blocks == 0)
            num_blocks++;
        dim3 threadsPerBlock(block_size, block_size);
        dim3 numBlocks(num_blocks, num_blocks);
        device_compute_residual<<<numBlocks, threadsPerBlock>>>(d_r, d_x, d_f, height, width, h_act);
        hipDeviceSynchronize();
    }

    static void
    ComputeRestriction(double *fine, double *coarse, int fine_N, int coarse_N)
    {
        int block_size = num_thread;
        int num_blocks = (coarse_N + block_size - 1) / block_size;
        if (num_blocks == 0)
            num_blocks++;
        dim3 threadsPerBlock(block_size, block_size);
        dim3 numBlocks(num_blocks, num_blocks);
        restriction_kernel_full_weighting<<<numBlocks, threadsPerBlock>>>(fine, coarse, fine_N, fine_N, coarse_N, coarse_N);
        hipDeviceSynchronize();
    }

    static void
    ComputeProlungator(double *coarse, double *fine, int coarse_N, int fine_N)
    {
        int block_size = num_thread;
        int num_blocks = fine_N / block_size;
        if (num_blocks == 0)
            num_blocks++;
        dim3 threadsPerBlock(block_size, block_size);
        dim3 numBlocks(num_blocks, num_blocks);
        prolungator_kernel<<<numBlocks, threadsPerBlock>>>(coarse, fine, coarse_N, coarse_N, fine_N, fine_N);
        hipDeviceSynchronize();
    }
};
