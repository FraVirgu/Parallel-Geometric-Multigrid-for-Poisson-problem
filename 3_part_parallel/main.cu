#include "ParallelTestRunner.cu"
#include "../2_part_MG/MultiGridTestRunner.hpp"
std::vector<int> N_list = {33, 65, 129, 257, 513, 1025, 2049};
std::vector<int> N_thread_list = {16, 32};
int main()
{
    // Allocate managed memory for temporary data
    double *tmp;
    hipMallocManaged(&tmp, sizeof(double) * 129);
    // Ensure memory is accessible on the GPU
    hipDeviceSynchronize();
    // Free the allocated memory after use
    hipFree(tmp);

    const int alpha = 3;
    const double epsilon = 1e-7;
    const int mg_max_iterations = 3;

    /*
    ParallelTestRunner parallel_runner(0, mg_max_iterations, alpha);
    // PLOT THE TIME DIFFERENCES BETWEEN ALL METHOD USED IN MG PARALLEL vs SEQUENTIAL
    parallel_runner.plotTimeSequentialVsParallel(N_list, N_thread_list);
    */

    /*
    // TIMINGS RESULTS

     // CPU
     MultigridTestRunner runner(epsilon, mg_max_iterations, alpha);
     runner.run_all_cycles_time_h(N_list);
   */
    // GPU
    ParallelTestRunner parallel_runner(0, mg_max_iterations, alpha);
    parallel_runner.run_all_cycles(N_list);

    // SAVE ERROR VECTOR
    /*

  // CPU
    N_list = {2049};
    MultigridTestRunner runner(epsilon, mg_max_iterations, alpha);
    runner.run_w_cycles_err_vector_iteration(N_list);

    // GPU
    N_list = {8193};
    parallel_runner.run_w_cycles_err_vector_iteration(N_list);
    */

    return 0;
}