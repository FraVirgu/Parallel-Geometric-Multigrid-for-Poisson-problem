#include "hip/hip_runtime.h"
#include "Parallel_Mg.cu"
#include "../Smoother.hpp"
#include "../DynamicGridUtils.hpp"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include "save_to_file.hpp"
// Restriction using full-weighting
void restrict_full_weighting_2(const double *fine, double *coarse, int Nf, int Nc)
{
    // Set Dirichlet boundaries on coarse grid (assume zero)
    for (int i = 0; i < Nc; ++i)
    {
        coarse[i] = 0.0;                 // Top row
        coarse[(Nc - 1) * Nc + i] = 0.0; // Bottom row
        coarse[i * Nc] = 0.0;            // Left column
        coarse[i * Nc + (Nc - 1)] = 0.0; // Right column
    }

    // Apply full-weighting to interior coarse grid points
    for (int jc = 1; jc < Nc - 1; ++jc)
    {
        for (int ic = 1; ic < Nc - 1; ++ic)
        {
            int i_f = 2 * ic;
            int j_f = 2 * jc;

            int idx_c = jc * Nc + ic;
            int idx_f = j_f * Nf + i_f;

            coarse[idx_c] = 0.25 * fine[idx_f] +
                            0.125 * (fine[idx_f + 1] + fine[idx_f - 1] +
                                     fine[idx_f + Nf] + fine[idx_f - Nf]) +
                            0.0625 * (fine[idx_f - Nf - 1] + fine[idx_f - Nf + 1] +
                                      fine[idx_f + Nf - 1] + fine[idx_f + Nf + 1]);
        }
    }
}

// Prolongation using bilinear interpolation
void prolongation_2(double *fine, const double *coarse, int Nf, int Nc)
{

    // Bilinear interpolation on interior points
    for (int jc = 1; jc < Nc - 1; ++jc)
    {
        for (int ic = 1; ic < Nc - 1; ++ic)
        {
            int i_f = 2 * ic;
            int j_f = 2 * jc;

            int idx_c = jc * Nc + ic;

            fine[j_f * Nf + i_f] += coarse[idx_c];
            fine[(j_f + 1) * Nf + i_f] += 0.5 * (coarse[idx_c] + coarse[idx_c + Nc]);
            fine[j_f * Nf + i_f + 1] += 0.5 * (coarse[idx_c] + coarse[idx_c + 1]);
            fine[(j_f + 1) * Nf + i_f + 1] += 0.25 * (coarse[idx_c] + coarse[idx_c + 1] +
                                                      coarse[idx_c + Nc] + coarse[idx_c + Nc + 1]);
        }
    }

    // Set Dirichlet boundary (zero) on the fine grid
    for (int j = 0; j < Nf; ++j)
    {
        for (int i = 0; i < Nf; ++i)
        {
            if (i == 0 || i == Nf - 1 || j == 0 || j == Nf - 1)
                fine[j * Nf + i] = 0.0;
        }
    }
}

using namespace std;
class ParallelTestRunner
{
public:
    int N;
    double epsilon = 1e-6;
    int alpha;
    int mg_max_iterations;
    std::vector<double> err_vec;

    std::vector<std::tuple<int, int, double>> time_residual_cpu;
    std::vector<std::tuple<int, int, double>> time_residual_gpu;

    std::vector<std::tuple<int, int, double>> time_jacobi_cpu;
    std::vector<std::tuple<int, int, double>> time_jacobi_gpu;

    std::vector<std::tuple<int, int, double>> time_restriction_cpu;
    std::vector<std::tuple<int, int, double>> time_restriction_gpu;

    std::vector<std::tuple<int, int, double>> time_prolungator_cpu;
    std::vector<std::tuple<int, int, double>> time_prolungator_gpu;

    ParallelTestRunner(int n, int mg_iterations, int alp) : N(n), alpha(alp), mg_max_iterations(mg_iterations) {};

    void plotTimeSequentialVsParallel(std::vector<int> N_list, std::vector<int> N_thread_list)
    {
        for (int n_thread : N_thread_list)
        {
            num_thread = n_thread;
            for (int n : N_list)
            {
                cout << "\t\tN: " << n << endl;
                this->N = n;
                this->run_all_methods();
            }
        }

        save_timings_to_file_all_methods(
            this->time_residual_cpu,
            this->time_residual_gpu,

            this->time_jacobi_cpu,
            this->time_jacobi_gpu,

            this->time_restriction_cpu,
            this->time_restriction_gpu,

            this->time_prolungator_cpu,
            this->time_prolungator_gpu

        );
    }

    void run_all_cycles(const std::vector<int> &N_list)
    {
        std::vector<std::pair<int, double>> time_parallel_V_cycle, time_parallel_W_cycle;

        for (int n : N_list)
        {
            this->N = n;
            std::cout << "\n=== GPU Multigrid Solution for N = " << N << " ===\n";
            auto time_v_cycle = run_v_cycle();
            auto time_w_cycle = run_w_cycle(false);
            time_parallel_V_cycle.push_back({N, time_v_cycle});
            time_parallel_W_cycle.push_back({N, time_w_cycle});
        }
        save_timings_to_file(time_parallel_V_cycle, time_parallel_W_cycle);
    }

    void run_w_cycles_err_vector_iteration(const std::vector<int> &N_list)
    {
        for (int n : N_list)
        {
            this->N = n;
            auto time_w_cycle = run_w_cycle(true);
        }
    }

    double run_v_cycle()
    {

        int L = N * N;
        double h = 1.0 / (N - 1);
        // Unified (device-accessible) memory
        double *phi;
        double *f;
        double *x_true = new double[L];
        size_t bytes = L * sizeof(double);
        hipMallocManaged(&phi, bytes);
        hipMallocManaged(&f, bytes);
        // hipDeviceSynchronize(); // Ensure memory is ready

        DynamicGridUtils::initialize_zeros(phi, L);
        DynamicGridUtils::compute_rhs(f, N, N, h); // Host version
        DynamicGridUtils::compute_exact_solution(x_true, h, N, N);
        ParallelMultiGridSolver parallel_mg_solver(alpha);

        auto start_v = std::chrono::high_resolution_clock::now();
        for (int iter = 0; iter < mg_max_iterations; iter++)
            parallel_mg_solver.v_cycle(phi, f, N, h);
        auto end_v = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_w = end_v - start_v;

        double *err = new double[L];

        DynamicGridUtils::compute_error(err, phi, x_true, L);
        double err_norm = DynamicGridUtils::norm(err, L);
        double x_true_norm = DynamicGridUtils::norm(x_true, L);
        double rel_err = err_norm / x_true_norm;
        cout << "  Final Relative L2 Error: " << rel_err << endl;
        std::cout << "  Elapsed Time: " << elapsed_w.count() << " seconds\n";
        return elapsed_w.count();
    }

    double run_w_cycle(bool err_vector)
    {
        int L = N * N;
        double h = 1.0 / (N - 1);
        // Unified (device-accessible) memory
        double *phi;
        double *f;
        double *x_true = new double[L];
        size_t bytes = L * sizeof(double);
        hipMallocManaged(&phi, bytes);
        hipMallocManaged(&f, bytes);
        // hipDeviceSynchronize(); // Ensure memory is ready

        DynamicGridUtils::initialize_zeros(phi, L);
        DynamicGridUtils::compute_rhs(f, N, N, h); // Host version
        DynamicGridUtils::compute_exact_solution(x_true, h, N, N);
        ParallelMultiGridSolver parallel_mg_solver(alpha);

        auto start_v = std::chrono::high_resolution_clock::now();
        for (int iter = 0; iter < mg_max_iterations; iter++)
            parallel_mg_solver.w_cycle(phi, f, N, h);
        auto end_v = std::chrono::high_resolution_clock::now();
        std::chrono::duration<double> elapsed_w = end_v - start_v;

        double *err = new double[L];

        DynamicGridUtils::compute_error(err, phi, x_true, L);
        if (err_vector)
        {
            this->err_vec.assign(err, err + L);
            std::vector<std::vector<double>> err_vect_iteration;
            err_vect_iteration.push_back(this->err_vec);
            save_errors_vector_to_file_last_iteration_gpu(err_vect_iteration);
        }
        double err_norm = DynamicGridUtils::norm(err, L);
        double x_true_norm = DynamicGridUtils::norm(x_true, L);
        double rel_err = err_norm / x_true_norm;
        cout << "  Final Relative L2 Error: " << rel_err << endl;
        std::cout << "  Elapsed Time: " << elapsed_w.count() << " seconds\n";
        return elapsed_w.count();
    }

private:
    void run_jacobi()
    {
        int L = N * N;
        double h = 1.0 / (N - 1);
        auto smoother = new JacobiSmoother(epsilon);
        // Host memory
        double *x_initial = new double[L];
        double *f = new double[L];
        double *res_host = new double[L];
        double *x_out = new double[L];

        // Unified (device-accessible) memory
        double *x_dev_initial;
        double *f_dev;
        double *res_dev;
        double *x_out_dev;

        size_t bytes = L * sizeof(double);
        hipMallocManaged(&x_dev_initial, bytes);
        hipMallocManaged(&f_dev, bytes);
        hipMallocManaged(&res_dev, bytes);
        hipMallocManaged(&x_out_dev, bytes);

        hipDeviceSynchronize(); // Ensure memory is ready

        // Init host and device arrays
        DynamicGridUtils::initialize_zeros(x_initial, L);
        hipDeviceSynchronize();                                        // Ensure previous device ops are done
        hipMemPrefetchAsync(x_dev_initial, bytes, hipCpuDeviceId, 0); // Prefetch to CPU
        hipDeviceSynchronize();                                        // Wait for prefetch to complete
        DynamicGridUtils::initialize_zeros(x_dev_initial, L);
        DynamicGridUtils::initialize_zeros(res_host, L);
        DynamicGridUtils::initialize_zeros(res_dev, L);
        DynamicGridUtils::initialize_zeros(x_out, L);
        DynamicGridUtils::initialize_zeros(x_out_dev, L);

        // Compute RHS on both host and device memory (they are the same in content)
        DynamicGridUtils::compute_rhs(f, N, N, h);     // Host version
        DynamicGridUtils::compute_rhs(f_dev, N, N, h); // Unified memory for device use

        if (N < 4096)
        {
            // Measure host-side Jacobi time
            auto start_cpu = std::chrono::high_resolution_clock::now();
            smoother->smooth(x_out, f, N, N, h, 100);
            auto end_cpu = std::chrono::high_resolution_clock::now();
            double time_cpu = std::chrono::duration<double>(end_cpu - start_cpu).count();
            time_jacobi_cpu.push_back({num_thread, N, time_cpu});
        }

        // Measure device-side Jacobi time
        Parallel device;
        auto start_gpu = std::chrono::high_resolution_clock::now();
        device.ComputeJacobi(x_dev_initial, f_dev, N, N, h, 100);
        hipDeviceSynchronize();
        auto end_gpu = std::chrono::high_resolution_clock::now();
        double time_gpu = std::chrono::duration<double>(end_gpu - start_gpu).count();
        time_jacobi_gpu.push_back({num_thread, N, time_gpu});

        // cout << DynamicGridUtils::compare_vector(x_dev_initial, x_out, L) << endl;

        // Clean up
        delete[] x_initial;
        delete[] f;
        delete[] res_host;
        hipFree(x_dev_initial);
        hipFree(f_dev);
        hipFree(res_dev);
    }
    void run_residual()
    {
        int L = N * N;
        double h = 1.0 / (N - 1);

        // Host memory
        double *x_initial = new double[L];
        double *f = new double[L];
        double *res_host = new double[L];

        // Unified (device-accessible) memory
        double *x_dev_initial;
        double *f_dev;
        double *res_dev;

        size_t bytes = L * sizeof(double);
        hipMallocManaged(&x_dev_initial, bytes);
        hipMallocManaged(&f_dev, bytes);
        hipMallocManaged(&res_dev, bytes);
        hipDeviceSynchronize(); // Ensure memory is ready

        // Init host and device arrays
        DynamicGridUtils::initialize_zeros(x_initial, L);
        hipDeviceSynchronize();                                        // Ensure previous device ops are done
        hipMemPrefetchAsync(x_dev_initial, bytes, hipCpuDeviceId, 0); // Prefetch to CPU
        hipDeviceSynchronize();                                        // Wait for prefetch to complete
        DynamicGridUtils::initialize_zeros(x_dev_initial, L);
        DynamicGridUtils::initialize_zeros(res_host, L);
        DynamicGridUtils::initialize_zeros(res_dev, L);

        // Compute RHS on both host and device memory (they are the same in content)
        DynamicGridUtils::compute_rhs(f, N, N, h);     // Host version
        DynamicGridUtils::compute_rhs(f_dev, N, N, h); // Unified memory for device use

        if (N < 4096)
        {
            // Measure host-side residual time
            auto start_cpu = std::chrono::high_resolution_clock::now();
            DynamicGridUtils::compute_residual(res_host, x_initial, f, N, N, h);
            auto end_cpu = std::chrono::high_resolution_clock::now();
            double time_cpu = std::chrono::duration<double>(end_cpu - start_cpu).count();
            time_residual_cpu.push_back({num_thread, N, time_cpu});
        }

        // Measure device-side residual time
        Parallel device;
        auto start_gpu = std::chrono::high_resolution_clock::now();
        device.ComputeResidual(res_dev, x_dev_initial, f_dev, N, N, h);
        hipDeviceSynchronize();
        auto end_gpu = std::chrono::high_resolution_clock::now();
        double time_gpu = std::chrono::duration<double>(end_gpu - start_gpu).count();
        time_residual_gpu.push_back({num_thread, N, time_gpu});

        // cout << DynamicGridUtils::compare_vector(res_dev, res_host, L) << endl;

        // Clean up
        delete[] x_initial;
        delete[] f;
        delete[] res_host;
        hipFree(x_dev_initial);
        hipFree(f_dev);
        hipFree(res_dev);
    }

    void run_restriction()
    {
        int L = N * N;
        int n_rest = N / 2;
        double h = 1.0 / (N - 1);
        double *f = new double[L];
        double *f_dev;

        int l_rest = n_rest * n_rest;
        double *output = new double[l_rest];
        double *output_dev;

        size_t bytes = L * sizeof(double);
        size_t bytes_restr = l_rest * sizeof(double);

        hipMallocManaged(&f_dev, bytes);
        hipMallocManaged(&output_dev, bytes_restr);

        DynamicGridUtils::initialize_zeros(f, L);
        DynamicGridUtils::initialize_zeros(f_dev, L);

        // Compute RHS on both host and device memory (they are the same in content)
        DynamicGridUtils::compute_rhs(f, N, N, h);     // Host version
        DynamicGridUtils::compute_rhs(f_dev, N, N, h); // Unified memory for device use

        if (N < 4096)
        {
            // Measure host-side restriction time
            auto start_cpu = std::chrono::high_resolution_clock::now();
            restrict_full_weighting_2(f, output, N, n_rest);
            auto end_cpu = std::chrono::high_resolution_clock::now();
            double time_cpu = std::chrono::duration<double>(end_cpu - start_cpu).count();
            time_restriction_cpu.push_back({num_thread, N, time_cpu});
        }

        // Measure device-side restriction time
        Parallel device;
        auto start_gpu = std::chrono::high_resolution_clock::now();
        device.ComputeRestriction(f_dev, output_dev, N, n_rest);
        hipDeviceSynchronize();
        auto end_gpu = std::chrono::high_resolution_clock::now();
        double time_gpu = std::chrono::duration<double>(end_gpu - start_gpu).count();
        time_restriction_gpu.push_back({num_thread, N, time_gpu});

        // cout << DynamicGridUtils::compare_vector(output_dev, output, l_rest) << endl;

        // Clean up
        delete[] f;
        delete[] output;
        hipFree(f_dev);
        hipFree(output_dev);
    }

    void run_prolungator()
    {
        int L = N * N;
        int n_prolungator = N * 2;
        double h = 1.0 / (N - 1);
        double *f = new double[L];
        double *f_dev;

        int l_prol = n_prolungator * n_prolungator;
        double *output = new double[l_prol];
        double *output_dev;

        size_t bytes = L * sizeof(double);
        size_t bytes_restr = l_prol * sizeof(double);

        DynamicGridUtils::initialize_zeros(f, L);
        DynamicGridUtils::initialize_zeros(output, l_prol);

        hipMallocManaged(&f_dev, bytes);
        hipMallocManaged(&output_dev, bytes_restr);
        DynamicGridUtils::initialize_zeros(output_dev, l_prol);
        DynamicGridUtils::initialize_zeros(f_dev, L);
        // Compute RHS on both host and device memory (they are the same in content)
        DynamicGridUtils::compute_rhs(f, N, N, h);     // Host version
        DynamicGridUtils::compute_rhs(f_dev, N, N, h); // Unified memory for device use

        if (N < 4096)
        {
            // Measure host-side prolongation time
            auto start_cpu = std::chrono::high_resolution_clock::now();
            prolongation_2(output, f, n_prolungator, N);
            auto end_cpu = std::chrono::high_resolution_clock::now();
            double time_cpu = std::chrono::duration<double>(end_cpu - start_cpu).count();
            time_prolungator_cpu.push_back({num_thread, N, time_cpu});
        }

        // Measure device-side prolongation time
        Parallel device;
        auto start_gpu = std::chrono::high_resolution_clock::now();
        device.ComputeProlungator(f_dev, output_dev, N, n_prolungator);
        hipDeviceSynchronize();
        auto end_gpu = std::chrono::high_resolution_clock::now();
        double time_gpu = std::chrono::duration<double>(end_gpu - start_gpu).count();
        time_prolungator_gpu.push_back({num_thread, N, time_gpu});
        // cout << DynamicGridUtils::compare_vector(output_dev, output, l_prol) << endl;

        // Clean up
        delete[] f;
        delete[] output;
        hipFree(f_dev);
        hipFree(output_dev);
    }

    void run_all_methods()
    {
        run_residual();
        run_jacobi();
        run_restriction();
        run_prolungator();
    }
};
