
#include <hip/hip_runtime.h>
__device__ void device_compute_residual(double *r, double *x, double *f, int height, int width, double h_act)
{
    int x_pos = blockIdx.x * blockDim.x + threadIdx.x;
    int y_pos = blockIdx.y * blockDim.y + threadIdx.y;
    int index = y_pos * width + x_pos;
    double r_out = 0.0;

    if (x_pos != 0 && y_pos != 0 && x_pos < width - 1 && y_pos < height - 1)
    {
        r_out = f[index] - (4 * x[index] - x[index - 1] - x[index + 1] - x[index - width] - x[index + width]) / (h_act * h_act);
    }
    __syncthreads();

    if (x_pos != 0 && y_pos != 0 && x_pos < width - 1 && y_pos < height - 1)
    {
        r[index] = r_out;
    }
    __syncthreads();
}
